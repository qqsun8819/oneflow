
#include <hip/hip_runtime.h>
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#ifdef WITH_CUDA
#include "oneflow/core/framework/framework.h"
#include "oneflow/user/kernels/ndindex_offset_util.h"

namespace oneflow {

namespace user_op {

template<typename T>
__global__ void OffsetToNdIndexForwardGpuKernel(int32_t dims_num, const T* offset, const T* dims,
                                                T* out) {
  DoOffsetToNdIndex<T>(dims_num, offset, dims, out);
}

template<typename T>
struct OffsetToNdIndexFunctor<DeviceType::kGPU, T> final {
  void operator()(DeviceCtx* ctx, int32_t dims_num, const T* offset, const T* dims, T* out) {
    RUN_CUDA_KERNEL((OffsetToNdIndexForwardGpuKernel<T>), ctx, dims_num, dims_num, offset, dims,
                    out);
  }
};

OF_PP_SEQ_PRODUCT_FOR_EACH_TUPLE(INSTANTIATE_OFFSET_TO_NDINDEX_FUNCTOR, (DeviceType::kGPU),
                                 NDINDEX_OFFSET_DATA_TYPE_SEQ);
}  // namespace user_op
}  // namespace oneflow

#endif  // End WITH_CUDA
