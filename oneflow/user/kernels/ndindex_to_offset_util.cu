
#include <hip/hip_runtime.h>
#ifdef WITH_CUDA
#include "oneflow/core/framework/framework.h"
#include "oneflow/user/kernels/ndindex_to_offset_util.h"


namespace oneflow {

namespace user_op {

template<typename T>
__global__ void IndexToOffsetForwardGpuKernel(int32_t in_num, 
  int32_t ndim, const T* index, const T* dims_tensor, T* out) {
  // printf("RUN CUDA KERNEL");
  DoIndexToOffset<T>(in_num, ndim, index, dims_tensor, out);
}

template<typename T>
struct NdIndexToOffsetFunctor<DeviceType::kGPU, T> final {
    void operator()(DeviceCtx* ctx, int32_t in_num,
        int32_t ndim, const T* index, const T* dims_tensor, T* out) {
    printf("Enter Cuda operator");
    RUN_CUDA_KERNEL((IndexToOffsetForwardGpuKernel<T>), ctx, in_num, in_num, ndim, index, dims_tensor, out);
  }
};

OF_PP_SEQ_PRODUCT_FOR_EACH_TUPLE(INSTANTIATE_NDINDEX_TO_OFFSET_FUNCTOR, (DeviceType::kGPU),
                                 NDINDEX_TO_OFFSET_DATA_TYPE_SEQ);
}  // namespace user_op
}  // namespace oneflow

#endif  // End WITH_CUDA